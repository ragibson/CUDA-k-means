
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define BLOCK_SIZE 512
#define MAX_POINTS 100000000 // 100M points
#define MAX_MEANS 1000
#define MAX_ITER 30

// CUDA prefers struct-of-arrays style here (for cache purposes)
typedef struct {
  double *x, *y;
  int *membership;
} points;

typedef struct {
  double *x, *y;
} centroids;

typedef struct {
  double *x_sum, *y_sum;
  int *size;
} temp_centroids;

// algorithm termination flag
__managed__ int assignment_changed = 1;

// reads n data points from input file
__host__ void read_data(int n, char *file_name, points P) {
  unsigned int i = 0;
  double x, y;
  FILE *file = fopen(file_name, "r");
  assert(file != NULL);

  while (!feof(file) && i < n) {
    if (fscanf(file, "%lf %lf", &x, &y) != 2)
      break;
    P.x[i] = x;
    P.y[i] = y;
    P.membership[i++] = -1;
  }
}

// selects k centers at random from n points
__host__ void init_centers(int n, int k, points P, centroids C) {
  srand(time(NULL));
  for (int i = 0; i < k; ++i) {
    // not actually uniform random sampling, but very close
    int rand_idx = rand() % n;
    C.x[i] = P.x[rand_idx];
    C.y[i] = P.y[rand_idx];
  }
}

// computes ||p-c||^2 for a point p and center c
__device__ inline double norm_2D_sqr(double x1, double y1,
                                     double x2, double y2) {
  // sqrt is monotonic, so we may omit it in the distance calculation
  // i.e. application of sqrt does not change the order of distances
  return (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2);
}

// assign each point to the cluster given by the closest centroid
// NVIDIA suggests const and restrict here to improve compiler optimization
__global__ void
assign_clusters(int n, int k,
                const double *__restrict__ Px,
                const double *__restrict__ Py,
                int *__restrict__ Pmembership,
                double *__restrict__ Cx,
                double *__restrict__ Cy,
                double *__restrict__ Ox_sum,
                double *__restrict__ Oy_sum,
                int *__restrict__ Osize) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  // thread-local values that will be reduced
  __shared__ double x_sum[BLOCK_SIZE];
  __shared__ double y_sum[BLOCK_SIZE];
  __shared__ int size[BLOCK_SIZE];

  int membership = -1;

  if (index < n) {
    double min_dist = INFINITY;
    for (int i = 0; i < k; ++i) {
      double current_dist = norm_2D_sqr(Px[index], Py[index], Cx[i], Cy[i]);
      if (current_dist < min_dist) {
        min_dist = current_dist;
        membership = i;
      }
    }

    // arbitrary concurrent write is valid since all
    // threads write the same value
    if (membership != Pmembership[index])
      assignment_changed = 1;
    Pmembership[index] = membership;
  }
  __syncthreads();

  // k reductions (one per centroid)
  for (int c = 0; c < k; ++c) {
    x_sum[tid] = (membership == c) ? Px[index] : 0;
    y_sum[tid] = (membership == c) ? Py[index] : 0;
    size[tid] = (membership == c) ? 1 : 0;
    __syncthreads();

    // reduce block's sums into one value (in thread 0)
    for (int offset = BLOCK_SIZE >> 1; offset > 0; offset >>= 1) {
      if (tid < offset) {
        x_sum[tid] += x_sum[tid + offset];
        y_sum[tid] += y_sum[tid + offset];
        size[tid] += size[tid + offset];
      }
      __syncthreads();
    }

    // save block's sums to output arrays
    if (tid == 0) {
      Ox_sum[blockIdx.x * k + c] = x_sum[tid];
      Oy_sum[blockIdx.x * k + c] = y_sum[tid];
      Osize[blockIdx.x * k + c] = size[tid];
    }
    __syncthreads();
  }
}

// reduce temporary cluster sizes and centroid x/y sums to smaller arrays
__global__ void
reduce_temp_clusters(int n, int k,
                     const double *__restrict__ Ix_sum,
                     const double *__restrict__ Iy_sum,
                     const int *__restrict__ Isize,
                     double *__restrict__ Ox_sum,
                     double *__restrict__ Oy_sum,
                     int *__restrict__ Osize) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int tid = threadIdx.x;

  // thread-local values that will be reduced
  __shared__ double x_sum[BLOCK_SIZE];
  __shared__ double y_sum[BLOCK_SIZE];
  __shared__ int size[BLOCK_SIZE];

  for (int c = 0; c < k; ++c) {
    x_sum[tid] = 0;
    y_sum[tid] = 0;
    size[tid] = 0;

    // if necessary, sum multiple items per thread
    for (int b = index; b < n; b += stride) {
      x_sum[tid] += Ix_sum[b * k + c];
      y_sum[tid] += Iy_sum[b * k + c];
      size[tid] += Isize[b * k + c];
    }
    __syncthreads();

    // reduce block's sums into one value (in thread 0)
    for (int offset = BLOCK_SIZE >> 1; offset > 0; offset >>= 1) {
      if (tid < offset) {
        x_sum[tid] += x_sum[tid + offset];
        y_sum[tid] += y_sum[tid + offset];
        size[tid] += size[tid + offset];
      }
      __syncthreads();
    }

    // save block's sums to output arrays
    if (tid == 0) {
      Ox_sum[blockIdx.x * k + c] = x_sum[tid];
      Oy_sum[blockIdx.x * k + c] = y_sum[tid];
      Osize[blockIdx.x * k + c] = size[tid];
    }
    __syncthreads();
  }
}

// update cluster centroid positions
__global__ void update_clusters(int n, int k,
                                double *__restrict__ Cx,
                                double *__restrict__ Cy,
                                const double *__restrict__ Ix_sum,
                                const double *__restrict__ Iy_sum,
                                const int *__restrict__ Isize) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < k && Isize[index]) {
    Cx[index] = Ix_sum[index] / Isize[index];
    Cy[index] = Iy_sum[index] / Isize[index];
  }
}

/*
 *  prints results and performance where
 *    k = number of clusters (means)
 *    n = number of points (in 2D)
 *    h = number of iterations until convergence
 *    t = elapsed time (in seconds)
 *
 *    P contains the input points
 *    C contains the final cluster centroids
 *    T contains (in part) the final cluster sizes
 */
__host__ void print_results(int k, int n, int h, double t,
                            points P, centroids C, temp_centroids T) {
  printf("performed %d iterations in %.2f s, perf: %.2f billion\n", h, t,
         (double)k * n * h / t * 1e-9);

  double *xs = (double *)malloc(sizeof(double) * n);
  double *ys = (double *)malloc(sizeof(double) * n);
  int offsets[k + 1];

  offsets[0] = 0;
  for (int i = 0; i < k; ++i) {
    offsets[i + 1] = offsets[i] + T.size[i];
  }

  // pack permutation of input points into clusters in a single pass by using
  // prefix-sum on the cluster sizes as offsets into our output arrays
  for (int i = 0; i < n; ++i) {
    int m = P.membership[i];
    xs[offsets[m]] = P.x[i];
    ys[offsets[m]++] = P.y[i];
  }

  for (int c = 0; c < k; ++c) {
    printf("=====cluster %d centered at %lf %lf has size %d=====\n", c, C.x[c],
           C.y[c], T.size[c]);
    for (int i = offsets[c] - T.size[c]; i < offsets[c]; ++i) {
      printf("%lf %lf\n", xs[i], ys[i]);
    }
  }

  free(xs);
  free(ys);
}

int main(int argc, char **argv) {
  int k, n, h;
  char *file_name;
  points P;
  centroids C;
  temp_centroids T1;
  temp_centroids T2;
  hipEvent_t start, stop;
  float time;

  // read in number of points and means
  assert(argc >= 4);
  n = atoi(argv[1]);
  k = atoi(argv[2]);
  file_name = argv[3];
  assert(n <= MAX_POINTS && k <= MAX_MEANS);

  int blockSize = BLOCK_SIZE;
  int numBlocks = (n + blockSize - 1) / blockSize;
  int reductionBlockSize = BLOCK_SIZE;
  int reductionNumBlocks =
      (numBlocks + reductionBlockSize - 1) / reductionBlockSize;

  // make sure that we can support the number of points with our two block
  // reductions. with BLOCK_SIZE = 512, this limit is ~250M points
  assert(reductionNumBlocks <= 1024);

  // malloc memory and set up GPU timers
  hipMallocManaged(&P.x, sizeof(double) * n);
  hipMallocManaged(&P.y, sizeof(double) * n);
  hipMallocManaged(&P.membership, sizeof(int) * n);
  hipMallocManaged(&C.x, sizeof(double) * k);
  hipMallocManaged(&C.y, sizeof(double) * k);
  hipMallocManaged(&T1.x_sum, sizeof(double) * numBlocks * k);
  hipMallocManaged(&T1.y_sum, sizeof(double) * numBlocks * k);
  hipMallocManaged(&T1.size, sizeof(int) * numBlocks * k);
  hipMallocManaged(&T2.x_sum, sizeof(double) * reductionNumBlocks * k);
  hipMallocManaged(&T2.y_sum, sizeof(double) * reductionNumBlocks * k);
  hipMallocManaged(&T2.size, sizeof(int) * reductionNumBlocks * k);
  hipEventCreate(&start);
  hipEventCreate(&stop);

  read_data(n, file_name, P);
  init_centers(n, k, P, C);

  hipEventRecord(start, 0);
  for (h = 0; assignment_changed && h < MAX_ITER; ++h) {
    // assign points to nearest clusters
    assignment_changed = 0;
    assign_clusters<<<numBlocks, blockSize>>>
      (n, k,
       P.x, P.y, P.membership,
       C.x, C.y,
       T1.x_sum, T1.y_sum, T1.size);
    hipDeviceSynchronize();

    // two block reductions of cluster sizes and centroid x/y sums
    reduce_temp_clusters<<<reductionNumBlocks, reductionBlockSize>>>
      (numBlocks, k,
       T1.x_sum, T1.y_sum, T1.size,  // input values to reduce
       T2.x_sum, T2.y_sum, T2.size); // reduced output values
    hipDeviceSynchronize();
    reduce_temp_clusters<<<1, reductionBlockSize>>>
      (reductionNumBlocks, k,
       T2.x_sum, T2.y_sum, T2.size,  // reduce values from T2
       T1.x_sum, T1.y_sum, T1.size); // back into T1
    hipDeviceSynchronize();

    // update centroid positions
    update_clusters<<<1, k>>>
      (n, k,
       C.x, C.y,
       T1.x_sum, T1.y_sum, T1.size);
    hipDeviceSynchronize();
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  print_results(k, n, h, time * 1e-3, P, C, T1);

  // CUDA automatically frees and resets device on program exit
}
